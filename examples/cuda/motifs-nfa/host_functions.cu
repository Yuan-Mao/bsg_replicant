#include "hip/hip_runtime.h"
// Original work:
// Copyright (C) 2010  
// Pierluigi Rolando (pierluigi.rolando@polito.it)
// Netgroup - DAUIN - Politecnico di Torino
//
// Niccolo' Cascarano (niccolo.cascarano@polito.it)
// Netgroup - DAUIN - Politecnico di Torino
//
// Modified work:
// Copyright (C) 2017  
// Vinh Dang (vqd8a@virginia.edu)
// University of Virginia
//
// This program is free software; you can redistribute it and/or
// modify it under the terms of the GNU General Public License
// as published by the Free Software Foundation; either version 2
// of the License, or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program; if not, write to the Free Software
// Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.

#include <cstdlib>
#include <cassert>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

#include <libgen.h>
#include <stdio.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>

#include "burst.h"
#include "common.h"
#include "cuda_allocator.h"
#include "host_functions.h"
#include "globals.h"
#include "gpu_functions.h"
#include "transition_graph.h"
#include "utils.h"
					
using namespace std;

/*--------------------------------------------------------------------------------------------------*/
#ifdef TEXTURE_MEM_USE //Texture memory: NFA STATE TABLE
texture<st_t, hipTextureType1D, hipReadModeElementType> tex_nfa_tables;
texture<st_t, hipTextureType1D, hipReadModeElementType> tex_src_tables;
texture<unsigned int, hipTextureType1D, hipReadModeElementType> tex_input_transition_tables;
__global__ void nfa_kernel_texture(	symbol_fetch *input,
									unsigned long *cur_size_vec,
									ST_BLOCK *_svs,// ST_BLOCK is just a fancy name for a chunk of 32-bit unsigned data
									unsigned int *st_vec_lengths,
									ST_BLOCK *persistents,
									unsigned int *match_count, match_type *match_array, unsigned int match_vec_size,
									unsigned int *accum_nfa_table_lengths, unsigned int *accum_offset_table_lengths, unsigned int *accum_state_vector_lengths);
#endif
/*--------------------------------------------------------------------------------------------------*/
void GPUMemInfo ()
{
   size_t free_byte ;
   size_t total_byte ;
   hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
   if ( hipSuccess != cuda_status ){   
      printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );   
      exit(1);   
   }
   double free_db = (double)free_byte ;
   double total_db = (double)total_byte ;
   double used_db = total_db - free_db ;

   printf("GPU memory usage: used = %lf MB, free = %lf MB, total = %f MB\n", used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
}
/*--------------------------------------------------------------------------------------------------*/
vector<set<unsigned> > nfa_execute(std::vector<TransitionGraph *> tg, Burst &burst, unsigned int n_subsets, 
#ifdef DEBUG
                                   int *rulestartvec,
#endif
                                   double *t_alloc, double *t_kernel, double *t_collect, int *blocksize, unsigned int *trans_per_sym, int blksiz_tuning){	

   struct timeval c0, c1, c2, c3, c33, c4;
   long seconds, useconds;
   unsigned int *h_match_count, *d_match_count;
   match_type   *h_match_array, *d_match_array;
   
   ofstream fp_report;
   char filename[200], bufftmp[10];
   
   st_t *d_nfa_tables, *d_src_tables;
   unsigned int *d_offset_tables;
   symbol *d_input;
   unsigned long *d_cur_size_vec;
   ST_BLOCK *d_svs, *d_persistents, *d_accepts;
   size_t max_shmem=0;
   unsigned int *accum_nfa_table_lengths, *accum_offset_table_lengths, *accum_state_vector_lengths, *st_vec_lengths;//Note: arrays contain accumulated values   
   unsigned int *d_accum_nfa_table_lengths, *d_accum_offset_table_lengths, *d_accum_state_vector_lengths, *d_st_vec_lengths;

//#ifdef DEBUG
	//cout << "------------- Preparing to launch kernel ---------------" << endl;
	//cout << "Packets (Streams or Number of CUDA blocks in x-dimension): " << burst.get_sizes().size() << endl;
	
	//cout << "Accumulated number of symbol per packet (stream): ";
	//for (int i = 0; i < burst.get_sizes().size(); i++)
	//	cout << burst.get_sizes()[i] << " ";
	//cout << endl;
    
	//cout << "Threads per block: " << cfg.get_threads_per_block() << endl;

	for (unsigned int i = 0; i < n_subsets; ++i) {//Changed
		cout << "Graph (NFA) " << i+1 << endl;
		//cout << "   + Shared memory size: " << cfg.get_state_vector(i).get_size()*2 << endl;
	    cout << "   + State count: " << cfg.get_state_count(i) << endl;
		cout << "   + State vector length: " << cfg.get_state_vector(i).get_size()/sizeof(ST_BLOCK) << endl;
 	    cout << endl;
		if (max_shmem <= cfg.get_state_vector(i).get_size()*2) max_shmem = cfg.get_state_vector(i).get_size()*2;
	}	
//#endif
    
	gettimeofday(&c0, NULL);
	
	unsigned int tmp_avg_count = burst.get_sizes()[burst.get_sizes().size()-1]/burst.get_sizes().size()*15/n_subsets;//just for now, size of each match array for each packet

	cout << "tmp_avg_count: "   << tmp_avg_count
         << ", n_packets: "     << burst.get_sizes().size() 
         << ", n_subsets: "     << n_subsets
         << ", Maximum matches allowed: " << (tmp_avg_count*burst.get_sizes().size()*n_subsets) << endl;
	
	h_match_array         = (match_type*)malloc ((tmp_avg_count*burst.get_sizes().size()) * n_subsets * sizeof(match_type));//just for now
	h_match_count         = (unsigned int*)malloc ((              burst.get_sizes().size()) * n_subsets * sizeof(unsigned int));//just for now 
	
    accum_nfa_table_lengths    = (unsigned int*)malloc (n_subsets * sizeof(unsigned int));
	accum_offset_table_lengths = (unsigned int*)malloc (n_subsets * sizeof(unsigned int));
	accum_state_vector_lengths = (unsigned int*)malloc (n_subsets * sizeof(unsigned int));
	st_vec_lengths             = (unsigned int*)malloc (n_subsets * sizeof(unsigned int));
	
	hipMalloc( (void **) &d_match_array,  (tmp_avg_count*burst.get_sizes().size()) * n_subsets * sizeof(match_type));//just for now
    hipMalloc( (void **) &d_match_count,  (              burst.get_sizes().size()) * n_subsets * sizeof(unsigned int));//just for now

	hipMalloc( (void **) &d_accum_nfa_table_lengths,    n_subsets * sizeof(unsigned int));
	hipMalloc( (void **) &d_accum_offset_table_lengths, n_subsets * sizeof(unsigned int));
	hipMalloc( (void **) &d_accum_state_vector_lengths, n_subsets * sizeof(unsigned int));
    hipMalloc( (void **) &d_st_vec_lengths,             n_subsets * sizeof(unsigned int));
	
	size_t tmp_nfa_table_total_size=0, tmp_offset_table_total_size=0;//in bytes
	size_t tmp_curr_nfa_table_size=0, tmp_curr_offset_table_size=0;//in bytes
	size_t tmp_accum_prev_nfa_table_size=0, tmp_accum_prev_offset_table_size=0;//in bytes
	size_t tmp_state_vector_total_size=0, tmp_curr_state_vector_size=0, tmp_accum_prev_state_vector_size=0;//in bytes
	
	for (unsigned int i = 0; i < n_subsets; i++) {//Find total size (in bytes) of each data structure
		tmp_nfa_table_total_size    +=  tg[i]->get_nfa_table_size();
		tmp_offset_table_total_size +=  tg[i]->get_offset_table_size();
		tmp_state_vector_total_size +=  tg[i]->get_mutable_persistent_states().get_size();
		st_vec_lengths[i]            =  cfg.get_state_vector(i).get_size()/sizeof(ST_BLOCK);
	}
	hipMalloc((void **) &d_nfa_tables,    tmp_nfa_table_total_size);//Allocate device memory
    hipMalloc((void **) &d_src_tables,    tmp_nfa_table_total_size);
	hipMalloc((void **) &d_offset_tables, tmp_offset_table_total_size);
	hipMalloc((void **) &d_persistents,   tmp_state_vector_total_size);
	hipMalloc((void **) &d_accepts,       tmp_state_vector_total_size);
	
	//GPUMemInfo();
	
	for (unsigned int i = 0; i < n_subsets; i++){//Copy to device memory
		hipError_t retval1, retval2, retval3, retval4, retval5;
		tmp_curr_nfa_table_size    =  tg[i]->get_nfa_table_size();
		tmp_curr_offset_table_size =  tg[i]->get_offset_table_size();
		tmp_curr_state_vector_size =  tg[i]->get_mutable_persistent_states().get_size();
		
		if (i==0){
			retval1 = hipMemcpy( d_nfa_tables,    tg[i]->get_nfa_table(),                                 tmp_curr_nfa_table_size,    hipMemcpyHostToDevice);
			retval2 = hipMemcpy( d_src_tables,    tg[i]->get_src_table(),                                 tmp_curr_nfa_table_size,    hipMemcpyHostToDevice);
			retval3 = hipMemcpy( d_offset_tables, tg[i]->get_offset_table(),                              tmp_curr_offset_table_size, hipMemcpyHostToDevice);
			retval4 = hipMemcpy( d_persistents,   tg[i]->get_mutable_persistent_states().get_host(false), tmp_curr_state_vector_size, hipMemcpyHostToDevice);
			retval5 = hipMemcpy( d_accepts,       tg[i]->get_accept_states().get_host(false),             tmp_curr_state_vector_size, hipMemcpyHostToDevice);
		}
		else{
			tmp_accum_prev_nfa_table_size    +=  tg[i-1]->get_nfa_table_size();
			tmp_accum_prev_offset_table_size +=  tg[i-1]->get_offset_table_size();
			tmp_accum_prev_state_vector_size +=  tg[i-1]->get_mutable_persistent_states().get_size();
	
			retval1 = hipMemcpy( &d_nfa_tables   [tmp_accum_prev_nfa_table_size/sizeof(st_t)],        tg[i]->get_nfa_table(),    tmp_curr_nfa_table_size,    hipMemcpyHostToDevice);
			retval2 = hipMemcpy( &d_src_tables   [tmp_accum_prev_nfa_table_size/sizeof(st_t)],        tg[i]->get_src_table(),    tmp_curr_nfa_table_size,    hipMemcpyHostToDevice);
			retval3 = hipMemcpy( &d_offset_tables[tmp_accum_prev_offset_table_size/sizeof(unsigned int)], tg[i]->get_offset_table(), tmp_curr_offset_table_size, hipMemcpyHostToDevice);
			retval4 = hipMemcpy( &d_persistents  [tmp_accum_prev_state_vector_size/sizeof(ST_BLOCK)],     tg[i]->get_mutable_persistent_states().get_host(false), tmp_curr_state_vector_size, hipMemcpyHostToDevice);
			retval5 = hipMemcpy( &d_accepts      [tmp_accum_prev_state_vector_size/sizeof(ST_BLOCK)],     tg[i]->get_accept_states().get_host(false),             tmp_curr_state_vector_size, hipMemcpyHostToDevice);
		}
		accum_nfa_table_lengths[i]    = tmp_accum_prev_nfa_table_size/sizeof(st_t);
        accum_offset_table_lengths[i] = tmp_accum_prev_offset_table_size/sizeof(unsigned int);
		accum_state_vector_lengths[i] = tmp_accum_prev_state_vector_size/sizeof(ST_BLOCK);
	
		CUDA_CHECK(retval1, "Error while copying nfa table to device memory");
		CUDA_CHECK(retval2, "Error while copying src table to device memory");
		CUDA_CHECK(retval3, "Error while copying offset table to device memory");
		CUDA_CHECK(retval4, "Error while copying persistent state vector to device memory");
		CUDA_CHECK(retval5, "Error while copying accepting state vector to device memory");
	}

	d_input = burst.get_d_payloads();
	d_cur_size_vec = burst.get_d_sizes();
	d_svs = burst.get_mutable_state_vectors_device();//Changed
	
	hipMemcpy( d_accum_nfa_table_lengths,    accum_nfa_table_lengths,    n_subsets * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy( d_accum_offset_table_lengths, accum_offset_table_lengths, n_subsets * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy( d_accum_state_vector_lengths, accum_state_vector_lengths, n_subsets * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy( d_st_vec_lengths,             st_vec_lengths,             n_subsets * sizeof(unsigned int), hipMemcpyHostToDevice);
	
	GPUMemInfo();
	
	//Theoretical occupancy calculation
	//Finding maximum number of transitions per symbol (character)
	unsigned int max_trans_max=0;
	unsigned int min_trans_min=4000000000;
	unsigned int avg_trans=0;
	for (unsigned int i = 0; i < n_subsets * (256+1); i++) { 
		if (max_trans_max <= trans_per_sym[i]) max_trans_max = trans_per_sym[i];
		if (min_trans_min >= trans_per_sym[i]) min_trans_min = trans_per_sym[i];
		avg_trans += trans_per_sym[i];
	}
	avg_trans = avg_trans/(n_subsets * (256+1));
	
	for (unsigned int j = 0; j < n_subsets; j++){
		unsigned int tmp_max_trans=0;
		unsigned int tmp_min_trans=4000000000;
		unsigned int tmp_avg_trans=0;
		for (unsigned int i = 0; i < (256+1); i++){ 
			if (tmp_max_trans <= trans_per_sym[j*(256+1)+i]) tmp_max_trans = trans_per_sym[j*(256+1)+i];
			if (tmp_min_trans >= trans_per_sym[j*(256+1)+i]) tmp_min_trans = trans_per_sym[j*(256+1)+i];
			tmp_avg_trans += trans_per_sym[j*(256+1)+i];
		}
		tmp_avg_trans = tmp_avg_trans/(256+1);
		//printf("Subset %d: max_trans = %d, min_trans = %d, avg_trans = %d\n", j, tmp_max_trans, tmp_min_trans, tmp_avg_trans);
	}
	//
		
	int device;
	hipDeviceProp_t props;
	hipGetDevice(&device);
	hipGetDeviceProperties(&props, device); printf("GPU MultiProcessors: %d\n", props.multiProcessorCount);
	
	int blockSize_init; // The launch configurator returned block size 
	int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
	int maxActiveBlocks_init;
	hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize_init, nfa_kernel, max_shmem, 0);
	hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks_init, nfa_kernel, blockSize_init, max_shmem);
	float occupancy_init = (maxActiveBlocks_init * blockSize_init / props.warpSize) / (float)(props.maxThreadsPerMultiProcessor / props.warpSize)*100;
	//printf("Initial theoretical GPU launch info: minGridSize = %d, blockSize_init = %d, maxActiveBlocks_init = %d, occupancy_init: %f\n", minGridSize, blockSize_init, maxActiveBlocks_init, occupancy_init);
	
	//blockSize_init found by hipOccupancyMaxPotentialBlockSize() sometimes is smaller than 1024 and sometimes can not give best performance as 1024 with small number of n_subsets and number of packets. So, we decided to overwrite with 1024 (hard coded, we know that 1024 can be accepted. An automatic code is needed in future) but still use the original occupancy_init.	
	if ( (burst.get_sizes().size()) * n_subsets <= props.multiProcessorCount)
		blockSize_init = 1024;
	
	int blockSize_ = blockSize_init;
	*blocksize  = blockSize_init;
	int maxActiveBlocks = maxActiveBlocks_init;
	float occupancy = occupancy_init;
	while (((maxActiveBlocks*props.multiProcessorCount < (burst.get_sizes().size()) * n_subsets) || (abs(occupancy - occupancy_init) > 5)) && 
	       (blockSize_>32)) {
		blockSize_-=32;
		hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, nfa_kernel, blockSize_, max_shmem);
		occupancy = (maxActiveBlocks * blockSize_ / props.warpSize) / (float)(props.maxThreadsPerMultiProcessor / props.warpSize)*100;
		//printf("Inter. theoretical GPU launch info: blockSize_ = %d, maxActiveBlocks = %d, occupancy: %f\n", blockSize_, maxActiveBlocks, occupancy);
		if (abs(occupancy - occupancy_init) <= 5){	
			*blocksize = blockSize_;
		}
	}
		//Calculate blocksize based on max_trans_max
		blockSize_ = blockSize_init;
		int blocksize_trans  = blockSize_init;
		maxActiveBlocks = maxActiveBlocks_init;
		occupancy = occupancy_init;
		while ((blockSize_ > max_trans_max) && (blockSize_>32)) {
			blockSize_-=32;
			hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, nfa_kernel, blockSize_, max_shmem);
			occupancy = (maxActiveBlocks * blockSize_ / props.warpSize) / (float)(props.maxThreadsPerMultiProcessor / props.warpSize)*100;
			//printf("Inter. theoretical GPU launch info (use max_trans_max): blockSize_ = %d, maxActiveBlocks = %d, occupancy: %f\n", blockSize_, maxActiveBlocks, occupancy);
			if ((abs(occupancy - occupancy_init) <= 5) && (blockSize_ > max_trans_max)){	
				blocksize_trans = blockSize_;
			}
		}
		hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, nfa_kernel, blocksize_trans, max_shmem);
		occupancy = (maxActiveBlocks * blocksize_trans / props.warpSize) / (float)(props.maxThreadsPerMultiProcessor / props.warpSize)*100;
		//printf("Final theoretical GPU launch info: blocksize_trans = %d, maxActiveBlocks = %d, occupancy: %f, max_trans_max = %d, min_trans_min = %d, avg_trans = %d\n", blocksize_trans, maxActiveBlocks, occupancy, max_trans_max, min_trans_min, avg_trans);
		//End: Calculate blocksize based on max_trans_max
	
	hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, nfa_kernel, *blocksize, max_shmem);
	occupancy = (maxActiveBlocks * (*blocksize) / props.warpSize) / (float)(props.maxThreadsPerMultiProcessor / props.warpSize)*100;
		
	//printf("Final theoretical GPU launch info: blocksize = %d, maxActiveBlocks = %d, occupancy: %f, max_trans_max = %d, min_trans_min = %d, avg_trans = %d\n", *blocksize, maxActiveBlocks, occupancy, max_trans_max, min_trans_min, avg_trans);
	
	if (*blocksize >= blocksize_trans) *blocksize = blocksize_trans;//find min		
	
	// Bind textures to d_nfa_tables, d_src_tables, d_offset_tables
#ifdef TEXTURE_MEM_USE
		hipChannelFormatDesc channelDesc1 = hipCreateChannelDesc<st_t>();
		hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<unsigned int>();
		hipBindTexture(0, tex_nfa_tables, d_nfa_tables, channelDesc1, tmp_nfa_table_total_size);
		hipBindTexture(0, tex_src_tables, d_src_tables, channelDesc1, tmp_nfa_table_total_size);
		hipBindTexture(0, tex_input_transition_tables, d_offset_tables, channelDesc2, tmp_offset_table_total_size);	
		printf("Texture memory usage: %.0lf bytes (%lf Mbytes)\n", (2*tmp_nfa_table_total_size + tmp_offset_table_total_size)/1.0, (2*tmp_nfa_table_total_size + tmp_offset_table_total_size)/1024.0/1024.0);
#endif
	
	gettimeofday(&c1, NULL);
	
	// Launch kernel (asynchronously)
	//printf("Size of symbol = %d, Size of unsigned char = %d\n",sizeof(symbol), sizeof(unsigned char));
	dim3 block(cfg.get_threads_per_block(),1);
	dim3 grid(burst.get_sizes().size(),n_subsets);
    
	if (blksiz_tuning == 1) {
		block.x = *blocksize;
		printf("Blocksize tuning is used!\n");
	}
	else {
		printf("Blocksize tuning is NOT used!\n");
	}	
	cout << "GPU launch info: block.x = " << block.x << ", grid.x = " << grid.x << ", grid.y = " << grid.y << ", shmem = " << max_shmem << endl;
    cout << "GPU kernel running ... " << endl;
#ifdef TEXTURE_MEM_USE
		printf("NFA STATE TABLE stored in texture memory!\n");
		nfa_kernel_texture<<<grid, block, max_shmem>>>(/* TODO: the next three are w/o reference! */
												(symbol_fetch*)d_input,
												d_cur_size_vec,
												d_svs,
												d_st_vec_lengths,
												d_persistents,
												d_match_count, d_match_array, tmp_avg_count,
												d_accum_nfa_table_lengths, d_accum_offset_table_lengths, d_accum_state_vector_lengths);
#else
		printf("NFA STATE TABLE stored in global memory!\n");
		nfa_kernel<<<grid, block, max_shmem>>>(/* TODO: the next three are w/o reference! */
												d_nfa_tables,
												d_src_tables,
												d_offset_tables,
												(symbol_fetch*)d_input,
												d_cur_size_vec,
												d_svs,
												d_st_vec_lengths,
												d_persistents,
												d_match_count, d_match_array, tmp_avg_count,
												d_accum_nfa_table_lengths, d_accum_offset_table_lengths, d_accum_state_vector_lengths);	
#endif
	
	hipDeviceSynchronize();
	
	gettimeofday(&c2, NULL);
	
    cout << "GPU kernel done!!!" << endl;

#ifdef TEXTURE_MEM_USE		
	// unbind textures from d_nfa_tables, d_src_tables, d_offset_tables
    hipUnbindTexture(tex_nfa_tables);
	hipUnbindTexture(tex_src_tables);
	hipUnbindTexture(tex_input_transition_tables);	
#endif
	
	//seconds  = c2.tv_sec  - c1.tv_sec;
	//useconds = c2.tv_usec - c1.tv_usec;
    //*t_kernel= ((double)seconds * 1000 + (double)useconds/1000.0);
	//printf("host_functions.cu: t_kernel= %lf(ms)\n", *t_kernel);
	
	hipMemcpy( h_match_count,  d_match_count,                 burst.get_sizes().size()  * n_subsets * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy( h_match_array, d_match_array, (tmp_avg_count*burst.get_sizes().size()) * n_subsets * sizeof(match_type), hipMemcpyDeviceToHost);

    gettimeofday(&c3, NULL);		

	// Collect results
	//Temporarily comment the following FOR loop
    printf("Collecting results and saving into files ...\n");
	unsigned int total_matches=0;
	for (unsigned int i = 0; i < n_subsets; i++) {
#ifdef TEXTURE_MEM_USE
		strcpy (filename,"Report_tex_");
#else
        strcpy (filename,"Report_global_");
#endif	
		snprintf(bufftmp, sizeof(bufftmp),"%d",n_subsets);
		strcat (filename,bufftmp);
		strcat (filename,"_");
		snprintf(bufftmp, sizeof(bufftmp),"%d",i+1);
		strcat (filename,bufftmp);
		strcat (filename,".txt");
		fp_report.open (filename); //cout << "Report filename:" << filename << endl;
		tg[i]->mapping_states2rules(&h_match_count[burst.get_sizes().size()*i], &h_match_array[tmp_avg_count*burst.get_sizes().size()*i], 
		                            tmp_avg_count, burst.get_sizes(), burst.get_padded_sizes(), fp_report
#ifdef DEBUG
									, rulestartvec, i 
#endif
									                );
		fp_report.close();
		for (unsigned int j = 0; j < burst.get_sizes().size(); j++)
			total_matches += h_match_count[j + burst.get_sizes().size()*i];
	}
	printf("Host - Total number of matches %d\n", total_matches);

    gettimeofday(&c33, NULL);

	vector<set<unsigned> >batch_accepted_rules;
    
	// Free some memory
	//d__sv = burst.get_mutable_state_vectors_device(), d_input = burst.get_d_payloads(), and  d_cur_size_vec = burst.get_d_sizes() are freed outside the host_functions.cu
	//tg->get_mutable_persistent_states().free_device();//free device mem for d_persistent = tg->get_mutable_persistent_states().get_device();
	
	hipFree(d_match_count);
	hipFree(d_match_array);
	hipFree(d_accum_nfa_table_lengths);
	hipFree(d_accum_offset_table_lengths);
	hipFree(d_accum_state_vector_lengths);
	hipFree(d_st_vec_lengths);
	hipFree(d_nfa_tables);
    hipFree(d_src_tables);
	hipFree(d_offset_tables);
	hipFree(d_persistents);
	hipFree(d_accepts);
	
	free(h_match_count);
	free(h_match_array);
	free(accum_nfa_table_lengths);
	free(accum_offset_table_lengths);
	free(accum_state_vector_lengths);
	free(st_vec_lengths);
	
	gettimeofday(&c4, NULL);
	
	seconds  = c1.tv_sec  - c0.tv_sec;
	useconds = c1.tv_usec - c0.tv_usec;
    *t_alloc = ((double)seconds * 1000 + (double)useconds/1000.0);
	
	seconds  = c2.tv_sec  - c1.tv_sec;
	useconds = c2.tv_usec - c1.tv_usec;
    *t_kernel= ((double)seconds * 1000 + (double)useconds/1000.0);
	
	seconds  = c3.tv_sec  - c2.tv_sec;
	useconds = c3.tv_usec - c2.tv_usec;
    *t_collect = ((double)seconds * 1000 + (double)useconds/1000.0);

	seconds  = c33.tv_sec  - c3.tv_sec;
	useconds = c33.tv_usec - c3.tv_usec;
	printf("host_functions.cu: t_postprocesscpu= %lf(ms)\n", ((double)seconds * 1000 + (double)useconds/1000.0));

	seconds  = c4.tv_sec  - c33.tv_sec;
	useconds = c4.tv_usec - c33.tv_usec;
	printf("host_functions.cu: t_free= %lf(ms)\n", ((double)seconds * 1000 + (double)useconds/1000.0));

	return batch_accepted_rules;
}
